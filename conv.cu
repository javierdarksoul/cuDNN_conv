#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
cv::Mat load_image(const char* image_path) {
  cv::Mat image = cv::imread(image_path, cv::IMREAD_COLOR);
  image.convertTo(image, CV_32FC3);
  cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
  return image;
}
void save_image(const char* output_filename,
                float* buffer,
                int height,
                int width) {
  cv::Mat output_image(height, width, CV_32FC3, buffer);
  // Make negative values zero.
  cv::threshold(output_image,
                output_image,
                /*threshold=*/0,
                /*maxval=*/0,
                cv::THRESH_TOZERO);
  cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
  output_image.convertTo(output_image, CV_8UC3);
  cv::imwrite(output_filename, output_image);
}


int main(int argc, char const *argv[]) {
  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  cv::Mat image = load_image("/home/javier/Imágenes/vino.jpg");
  //save_image("test.jpg",image,image.rows,image.colums);
  //descriptor de entrada


  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NHWC,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/1,
                                        /*channels=*/3,
                                        /*image_height=*/image.rows,
                                        /*image_width=*/image.cols));

  //output descriptor
  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NHWC,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/1,
                                        /*channels=*/3,
                                        /*image_height=*/image.rows,
                                        /*image_width=*/image.cols));

  //descriptor del kernel
  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*format=*/HIPDNN_TENSOR_NCHW,
                                      /*out_channels=*/3,
                                      /*in_channels=*/3,
                                      /*kernel_height=*/3,
                                      /*kernel_width=*/3));

  //descriptor de la convolucion
  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                           /*pad_height=*/1,
                                           /*pad_width=*/1,
                                           /*vertical_stride=*/1,
                                           /*horizontal_stride=*/1,
                                           /*dilation_height=*/1,
                                           /*dilation_width=*/1,
                                           /*mode=*/HIPDNN_CROSS_CORRELATION,
                                           /*computeType=*/HIPDNN_DATA_FLOAT));

                                           
  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(
      hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                          /*memoryLimitInBytes=*/0,
                                          &convolution_algorithm)); 
 
  size_t workspace_bytes = 0;
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   convolution_descriptor,
                                                   output_descriptor,
                                                   convolution_algorithm,
                                                   &workspace_bytes));
  std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
          << std::endl;
  

//asignacion de memoria
void* d_workspace{nullptr};
hipMalloc(&d_workspace, workspace_bytes);
int image_bytes = 1 * 3 * image.rows * image.cols * sizeof(float);
float* d_input{nullptr};
hipMalloc(&d_input, image_bytes);
hipMemcpy(d_input, image.ptr<float>(0), image_bytes, hipMemcpyHostToDevice);
float* d_output{nullptr};
hipMalloc(&d_output, image_bytes);
hipMemset(d_output, 0, image_bytes);

//definimos el kernel
const float kernel_template[3][3] = {
  {1,  1, 1},
  {1, -8, 1},
  {1,  1, 1}
};

float h_kernel[3][3][3][3];
for (int kernel = 0; kernel < 3; ++kernel) {
  for (int channel = 0; channel < 3; ++channel) {
    for (int row = 0; row < 3; ++row) {
      for (int column = 0; column < 3; ++column) {
        h_kernel[kernel][channel][row][column] = kernel_template[row][column];
      }
    }
  }
}

float* d_kernel{nullptr};
hipMalloc(&d_kernel, sizeof(h_kernel));
hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

const float alpha = 1, beta = 1;


// convolution forward
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start);
checkCUDNN(hipdnnConvolutionForward(cudnn,
                                  &alpha,
                                  input_descriptor,
                                  d_input,
                                  kernel_descriptor,
                                  d_kernel,
                                  convolution_descriptor,
                                  convolution_algorithm,
                                  d_workspace,
                                  workspace_bytes,
                                  &beta,
                                  output_descriptor,
                                  d_output));
hipEventRecord(stop);
hipEventSynchronize(stop);
float milliseconds = 0;
hipEventElapsedTime(&milliseconds, start, stop);
printf("%f\n",milliseconds/(3.0f*1000.0f));

//save image
float* h_output = new float[image_bytes];
hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);
save_image("output_gpu.png", h_output, image.rows, image.cols);
//backward




}